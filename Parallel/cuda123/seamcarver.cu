#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "seamcarver.h"
#include <conio.h>

#define MAX_THREADS 1024
using namespace std;

// Kernel functions.

__global__
static void find_min_kernel(float* row, float* mins, int* min_indices,
    int width, int power) {
  // Compute current index.
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int index = tx + bx * power;

  // Set up shared memory for tracking mins.
  extern __shared__ float shared_memory[];
  float* shared_mins = (float*) shared_memory;
  int* shared_min_indices = (int*) (&(shared_memory[power]));

  // Copy global intermediate values into shared memory.
  shared_mins[tx] = (index < width) ? row[index] : MAX_VALUE;
  shared_min_indices[tx] = (index < width) ? index : MAX_VALUE;
  
  __syncthreads();

  // Do the reduction for value pairs.
  for (int i = power / 2; i > 0; i >>= 1){
		if (tx < i)
				if (shared_mins[tx] > shared_mins[tx + i])
				{
					shared_mins[tx] = shared_mins[tx + i];
					shared_min_indices[tx] = shared_min_indices[tx + i];
				}
		__syncthreads();
  }

  // Thread 0 has the solution.
	if (tx == 0){
			mins[bx] = shared_mins[0];
			min_indices[bx] = shared_min_indices[0];
	}
}


__global__
void compute_min_cost_kernel(float* energies, float* min_costs , float* temp_d ,
    int width, int height, int row) {
  // Extract thread and block index information
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int col = bx * MAX_THREADS + tx;

	if(col >= width )  // for excess threads
		  return;

	float left , right , middle;
	if(bx == 0)
		  left = (tx > 0) ? temp_d[tx - 1] : MAX_VALUE;
	else
		  left = temp_d[col - 1];
	middle = temp_d[col];
	right = ( col < width - 1) ? temp_d[col + 1] : MAX_VALUE;

	float minimum = min(left, min(middle, right));
    float cost= minimum + energies[row * width + col];
	
    __syncthreads();
    temp_d[col] = cost;

   	__syncthreads();
	min_costs[row * width + col] = cost;
}

// Class methods.


Seamcarver::Seamcarver(Image* image) {
  _image = image;
}

Seamcarver::~Seamcarver() {

}

// Simply remove n seams.
void Seamcarver::removeSeams(int n) {
  for (int i = 0; i < n; i++) 
		removeSeam();
}

// Removes 1 seam.
void Seamcarver::removeSeam() {
	findSeam();
	_image->removeSeam(_seam);
}


// Finds the seam of the lowest cost.
void Seamcarver::findSeam() {
  Energies energies(_image);
  energies.compute();
  float* energies_h = energies.getEnergies();

  int width = energies.width();
  int height = energies.height();

  // Declare pointers for device and host memory
  float* energies_d;
  float* min_cost_d;
  float* temp_d;
  float* temp_h = &(energies_h[0]);
    
  int row_size= width * sizeof(float);
  int size = width * height * sizeof(float);
  
  // Allocate device memory and for inputs and outputs
  hipMalloc((void**) &energies_d, size);
  hipMemcpy(energies_d, energies_h, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &min_cost_d, size);
  hipMalloc((void**) &temp_d,row_size);
  hipMemcpy(temp_d, temp_h, row_size, hipMemcpyHostToDevice);
 
  // Invoke the kernel to compute the min cost table
  int num_blocks = (width - 1) / MAX_THREADS + 1;
  int num_threads = min(MAX_THREADS, width);
  dim3 dim_grid(num_blocks, 1, 1);
  dim3 dim_block(num_threads, 1, 1);
  
  for(int row = 1; row < height ; row++)        // calculate minimum cost table row by row
		 compute_min_cost_kernel<<<dim_grid, dim_block>>>
			(energies_d, min_cost_d,temp_d, width, height, row);          //kernel call happens height times ie no. of rows

  // Transfer result from device to host
  hipMemcpy(energies_h, min_cost_d, size, hipMemcpyDeviceToHost);
  hipFree(energies_d);
  hipFree(min_cost_d);

   // Calculate threads and blocks for a minimum reduction
  num_threads = min(nextPower2(width), MAX_THREADS); // nextPower2 For nearest Power of 2 ie for 1029 ans. 2048
  num_blocks = (width - 1) / num_threads + 1;
  int mins_size = num_blocks * sizeof(float);
  int min_indices_size = num_blocks * sizeof(int);
  int shared_size = num_threads * (sizeof(float) + sizeof(int));

  // Declare pointers for device and host memory
  float* row = &(energies_h[(height - 1) * width]);
  float* mins = (float*) malloc(mins_size);
  int* min_indices = (int*) malloc(min_indices_size);
  float* row_d;
  float* mins_d;
  int* min_indices_d;

  hipMalloc((void**) &row_d, row_size);
  hipMemcpy(row_d, row, row_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &mins_d, mins_size);
  hipMalloc((void**) &min_indices_d, mins_size);
  
  // Use the kernel function to find intermediate minimums
  find_min_kernel<<<num_blocks, num_threads, shared_size>>>
      (row_d, mins_d, min_indices_d, width, num_threads);

  // Compute final minimum
  hipMemcpy(mins, mins_d, mins_size, hipMemcpyDeviceToHost);
  hipMemcpy(min_indices, min_indices_d, min_indices_size,
	hipMemcpyDeviceToHost);
  float minimum = mins[0];
  int min_index = min_indices[0];
   // finding minimum from bottom row
   for (int i = 1; i < num_blocks; i++) {
    if (mins[i] < minimum) {
		minimum = mins[i];
		min_index = min_indices[i];
    }
 }
   
  // Create the seam in reverse order.
  _seam.clear();
  _seam.push_back(min_index);

  for (int i = height - 2; i >= 0; i--) {
    float left = energies.get(i, min_index - 1);
    float middle = energies.get(i, min_index);
    float right = energies.get(i, min_index + 1);

    // Have the seam follow the least cost.
    if (left < middle && left < right) {
      min_index--; // go left
    } else if (right < middle && right < left) {
      min_index++; // go right
    }
	// Append to the seam.
    _seam.push_back(min_index);
  }

  // Clean up.
  std::reverse(_seam.begin(), _seam.end());
}

int Seamcarver::nextPower2(int n) {
  n--;
  n = n >>  1 | n;
  n = n >>  2 | n;
  n = n >>  4 | n;
  n = n >>  8 | n;
  n = n >> 16 | n;
  return ++n;
}
