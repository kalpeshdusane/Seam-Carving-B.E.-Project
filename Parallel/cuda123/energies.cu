#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "energies.h"

#define BLOCK_WIDTH 32
#define BLOCK_HEIGHT 32

using std::cout;
using std::endl;
using std::min;


// Kernel functions.

__global__
void compute_energies_kernel(RGBQuad* image, float* energies,
    int width, int height) {
  // Extract thread and block index information
  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;
  
  int row = by * BLOCK_HEIGHT + ty;
  int col = bx * BLOCK_WIDTH + tx;
  
  // Allocate shared memory with padding to avoid bank conflicts
  __shared__ RGBQuad shared_image[BLOCK_HEIGHT * (BLOCK_WIDTH + 1)];

  // Populate shared memory with image pixels
  if (row < height && col < width) {
		shared_image[ty * (blockDim.x + 1) + tx] = image[row * width + col];
  } else {
		RGBQuad& pixel = shared_image[ty * (BLOCK_WIDTH + 1) + tx];
		pixel.red = 0;
		pixel.green = 0;
		pixel.blue = 0;
		return;
  }

  // Wait for all threads to finish loading shared memory
  __syncthreads();

  // Compute differences with boundary checks
  RGBQuad& current = shared_image[ty * (BLOCK_WIDTH + 1) + tx];
  float dx2;
  float dy2;
  float dx_red,dx_green,dx_blue;
  float dy_red,dy_green,dy_blue;
  // Edges of blocks and image are special cases.
  if (col == width - 1) {
     dx_red = (float)current.red;
     dx_green = (float)current.green;
     dx_blue = (float)current.blue;
  }
  else if (tx == BLOCK_WIDTH - 1) {
     RGBQuad& right = image[row * width + col + 1]; // Access Global memory for Boundary Block
     dx_red = (float)right.red - current.red;
     dx_green = (float)right.green - current.green;
     dx_blue = (float)right.blue - current.blue;
  }else {
     RGBQuad& right = shared_image[ty * (BLOCK_WIDTH + 1) + tx + 1];
     dx_red = (float)right.red - current.red;
     dx_green = (float)right.green - current.green;
     dx_blue = (float)right.blue - current.blue;
  }
      // compute gradient in x-direction
  dx2 = dx_red * dx_red + dx_green * dx_green + dx_blue * dx_blue;

  if (row == height - 1) {
     dy_red = (float)current.red;
     dy_green = (float)current.green;
     dy_blue = (float)current.blue;
  }
  else if (ty == BLOCK_HEIGHT - 1) {
     RGBQuad& down = image[(row + 1) * width + col];		//Access Global Memory
     dy_red = (float)down.red - current.red;
     dy_green = (float)down.green - current.green;
     dy_blue = (float)down.blue - current.blue;
  }else {
     RGBQuad& down = shared_image[(ty + 1) * (BLOCK_WIDTH + 1) + tx];
     dy_red = (float)down.red - current.red;
     dy_green = (float)down.green - current.green;
     dy_blue = (float)down.blue - current.blue;
  }
		//compute gradient in y-direction
  dy2 = dy_red * dy_red + dy_green * dy_green + dy_blue * dy_blue;

  // Compute gradient
  float grad = (float) sqrt(dx2 + dy2);

  // Store results
  int index = row * width + col;
  energies[index] = grad;
}

// Class methods.

Energies::Energies(Image* image) {
  _width = image->width();
  _height = image->height();
  _image = image;
  _energies = new float[_width * _height];
}

Energies::~Energies() {
  delete _energies;
}

// Filtering options

void Energies::compute() {
  // Declare pointers for device memory
  RGBQuad* image_d;
  float* energies_d;
  int image_size = _width * _height * sizeof(RGBQuad);
  int energies_size = _width * _height * sizeof(float);

  // Allocate device memory and for inputs and outputs
  hipMalloc((void**) &image_d, image_size);
  hipMemcpy(image_d, _image->getPixels(), image_size, hipMemcpyHostToDevice);
  hipMalloc((void**) &energies_d, energies_size);

  // Invoke the kernel to compute the energies
  int num_blocks_x = (_width - 1) / BLOCK_WIDTH + 1;
  int num_blocks_y = (_height - 1) / BLOCK_HEIGHT + 1;
  int num_threads_x = min(BLOCK_WIDTH, _width);
  int num_threads_y = min(BLOCK_HEIGHT, _height);
  dim3 dim_grid(num_blocks_x, num_blocks_y, 1);
  dim3 dim_block(num_threads_x, num_threads_y, 1);
			//kernel call
  compute_energies_kernel<<<dim_grid, dim_block>>>
      (image_d, energies_d, _width, _height);

  // Transfer result from device to host
  hipMemcpy(_energies, energies_d, energies_size, hipMemcpyDeviceToHost);

  hipFree(image_d);
  hipFree(energies_d);
}

// Getters and operators

int Energies::width() const {
  return _width;
}

int Energies::height() const {
  return _height;
}

float Energies::get(int row, int col) const {
  if (row < 0 || row >= _height ||
      col < 0 || col >= _width) {
    // Return a high value so this is never the min.
    return MAX_VALUE;
  }

  // Common case.
  int index = row * _width + col;
  return _energies[index];
}

float* Energies::getEnergies() const {
  return _energies;
}

void Energies::set(int row, int col, float value) {
  int index = row * _width + col;
  _energies[index] = value;
}

void Energies::print() const {
  cout << "Energies: " << endl;
  for (int i = 0; i < _height; i++) {
    for (int j = 0; j < _width; j++) {
      cout << get(i, j) << "\t";
    }
    cout << endl;
  }
  cout << endl;
}

const float* Energies::operator [](int i) const {
  return _energies + (i * _width);
};
